#include "hip/hip_runtime.h"
#include "GpuSimulation.cuh"
#include "Lattice.hpp"
#include <iostream>
#include <omp.h>

int __calculateBoundary(const std::vector<int> &inputBoundary)
{
    // calculate host boundary with streaming conventions.
    // 0 is no boundary, 1 is right, 2 is up, 3 is left, 4 is down, 5 is up-right, 6 is up-left, 7 is down-left, 8 is
    // down-right

    if (inputBoundary.at(0) == 0 && inputBoundary.at(1) == 0)
        return 0;
    else if (inputBoundary.at(0) == 1 && inputBoundary.at(1) == 0)
        return 1;
    else if (inputBoundary.at(0) == 0 && inputBoundary.at(1) == -1)
        return 2;
    else if (inputBoundary.at(0) == -1 && inputBoundary.at(1) == 0)
        return 3;
    else if (inputBoundary.at(0) == 0 && inputBoundary.at(1) == 1)
        return 4;
    else if (inputBoundary.at(0) == 1 && inputBoundary.at(1) == -1)
        return 5;
    else if (inputBoundary.at(0) == -1 && inputBoundary.at(1) == -1)
        return 6;
    else if (inputBoundary.at(0) == -1 && inputBoundary.at(1) == 1)
        return 7;
    else if (inputBoundary.at(0) == 1 && inputBoundary.at(1) == 1)
        return 8;
    else
        return -1;
}

Lattice::Lattice(const std::string &filename)
{
    // Read the lattice from the file
    std::ifstream file;
    file.open(filename);
    if (!file.is_open())
    {
        throw std::runtime_error("Could not open file");
    }

    // read type of problem
    file >> problemType;
    file.get(); // Skip the newline

    // Read the number of cells in each dimension until newline
    std::vector<int> shape;
    int dimensions = 0;
    while (file.peek() != '\n')
    {
        int numCells;
        file >> numCells;
        shape.push_back(numCells);
        ++dimensions;
    }
    if (dimensions == 2)
    {
        structure = Structure::D2Q9;
    }
    /*
    else if (dimensions == 3)
    {
        structure = Structure::D3Q27;
    }
    */
    else
    {
        throw std::runtime_error("Invalid number of dimensions");
    }
    file.get(); // Skip the newline

    // Read Reynolds number
    float reynolds;
    file >> reynolds;

    // Read the simulation time
    float simulationTime;
    file >> simulationTime;

    if (problemType == 1)
    {
        file >> uLid;
    }
    file.get(); // Skip the newline

    // calculate simulation parameters
    sigma = 10.0 * shape.at(0);
    omP = 1.0 / (0.5 + 3.0 * uLid * shape.at(0) / reynolds);
    omM = 1.0 / (1.0 / (12.0 * uLid * shape.at(0) / reynolds) + 0.5);
    maxIt = (int)std::round(simulationTime * shape.at(0) / uLid);

    // Initialize the cells
    cells = NDimensionalMatrix<Cell>(shape);

    // Read the obstacles : for each newline, read the coordinates of the obstacle
    NDimensionalMatrix<bool> obstacles(shape);
    for (int i = 0; i < obstacles.getTotalSize(); ++i)
    {
        obstacles.setElementAtFlatIndex(i, false);
    }
    while (file.peek() != EOF)
    {
        std::vector<int> indices;
        for (int i = 0; i < dimensions; ++i)
        {
            int index;
            file >> index;
            indices.push_back(index);
        }
        obstacles.setElement(indices, true);
        file.get(); // Skip the newline
    }

    //  Initialize the cells one by one
    std::vector<float> f;
    std::vector<int> boundary;
    std::vector<int> indices;
    for (int i = 0; i < cells.getTotalSize(); ++i)
    {
        indices = cells.getIndicesAtFlatIndex(i);
        const bool &obstacle = obstacles.getElementAtFlatIndex(i);

        boundary.clear();
        f.clear();

        for (int k = 0; k < dimensions; ++k)
        {
            const int indexOfCurrDimension = indices.at(k);
            const int lenghtOfCurrDimension = shape.at(k);
            if (indexOfCurrDimension == 0)
            {
                boundary.push_back(-1);
            }
            else if (indexOfCurrDimension == lenghtOfCurrDimension - 1)
            {
                boundary.push_back(1);
            }
            else
            {
                boundary.push_back(0);
            }

            /*
            // check if there is an obstacle in the adjacent cell
            if (indexOfCurrDimension > 0)
            {
                std::vector<int> adjacentIndices = indices;
                adjacentIndices.at(i) -= 1;
                if (obstacles.getElement(adjacentIndices))
                {
                    boundary.at(i) = -1;
                }
            }

            if (indexOfCurrDimension < lenghtOfCurrDimension - 1)
            {
                std::vector<int> adjacentIndices = indices;
                adjacentIndices.at(i) += 1;
                if (obstacles.getElement(adjacentIndices))
                {
                    boundary.at(i) = 1;
                }
            }
            */
        }

        // f is 1
        for (int j = 0; j < structure.velocity_directions; ++j)
        {
            f.push_back(1);
        }

        cells.setElementAtFlatIndex(i, Cell(structure, boundary, obstacle, f));
    }

    // Close the file
    file.close();
}

void Lattice::simulate(std::ofstream &file)
{
    const float temp = 2.0 * sigma * sigma;
    const float halfOmpOmmSub = 0.5 * (omP - omM);
    const float halfOmpOmmSum = 0.5 * (omP + omM);
    while (timeInstant <= maxIt)
    {
        const float uLidNow = uLid * (1.0 - std::exp(-static_cast<double>(timeInstant * timeInstant) / temp));
// update cells
#pragma omp parallel
        {
#pragma omp for
            for (int j = 0; j < cells.getTotalSize(); ++j)
            {
                if (timeInstant != 0)
                {
                    cells.getElementAtFlatIndex(j).zouHe();
                }
                cells.getElementAtFlatIndex(j).updateMacro(structure);
                cells.getElementAtFlatIndex(j).setInlets(structure, uLidNow, problemType);
                cells.getElementAtFlatIndex(j).equilibriumCollision(structure, omP, halfOmpOmmSum, halfOmpOmmSub);
            }
#pragma omp for
            for (int j = 0; j < cells.getTotalSize(); ++j)
            {
                cells.getElementAtFlatIndex(j).streaming(*this, cells.getIndicesAtFlatIndex(j));
            }
        }

        // write to file every maxIt/100 time steps
        if (timeInstant % (maxIt / 100) == 0)
        {
            // write to file time instant
            file << timeInstant << '\n';

            // loop dimensions
            for (int i = 0; i < structure.dimensions; ++i)
            {
                // write to file macroU
                for (int j = 0; j < cells.getTotalSize(); ++j)
                {
                    file << cells.getElementAtFlatIndex(j).getMacroU().at(i) << ' ';
                }
                file << '\n';
            }
            // print to console every 100 time steps
            std::cout << "Time step: " << timeInstant << '\n';
        }

        // advance time
        timeInstant++;
    }
}

/// @brief supports only 2D lattice
void Lattice::simulateGpu(std::ofstream &file)
{
    const int nx = cells.getShape().at(0);
    const int ny = cells.getShape().at(1);

    // device allocations

    float *host_f, *host_new_f, *host_rho, *host_ux, *host_uy, *dev_f, *dev_new_f, *dev_rho, *dev_ux, *dev_uy;
    int *host_boundary, *dev_boundary;
    bool *host_obstacle, *dev_obstacle;

    // allocate memory on device
    hipMalloc((void **)&dev_f, cells.getTotalSize() * 9 * sizeof(float));
    hipMalloc((void **)&dev_new_f, cells.getTotalSize() * 9 * sizeof(float));
    hipMalloc((void **)&dev_rho, cells.getTotalSize() * sizeof(float));
    hipMalloc((void **)&dev_ux, cells.getTotalSize() * sizeof(float));
    hipMalloc((void **)&dev_uy, cells.getTotalSize() * sizeof(float));
    hipMalloc((void **)&dev_boundary, cells.getTotalSize() * sizeof(int));
    hipMalloc((void **)&dev_obstacle, cells.getTotalSize() * sizeof(bool));

    // allocate memory on host
    hipHostMalloc((void **)&host_f, cells.getTotalSize() * 9 * sizeof(float));
    hipHostMalloc((void **)&host_new_f, cells.getTotalSize() * 9 * sizeof(float));
    hipHostMalloc((void **)&host_rho, cells.getTotalSize() * sizeof(float));
    hipHostMalloc((void **)&host_ux, cells.getTotalSize() * sizeof(float));
    hipHostMalloc((void **)&host_uy, cells.getTotalSize() * sizeof(float));
    hipHostMalloc((void **)&host_boundary, cells.getTotalSize() * sizeof(int));
    hipHostMalloc((void **)&host_obstacle, cells.getTotalSize() * sizeof(bool));

#pragma omp parallel for
    // set host data
    for (int i = 0; i < cells.getTotalSize(); ++i)
    {
        const Cell &cell = cells.getElementAtFlatIndex(i);
        const std::vector<float> &f = cell.getF();
        const std::vector<float> &new_f = cell.getNewF();
        const std::vector<float> &macroU = cell.getMacroU();
        const std::vector<int> &boundary = cell.getBoundary();
        const bool &obstacle = cell.isObstacle();

        for (int j = 0; j < 9; ++j)
        {
            host_f[i * 9 + j] = f.at(j);
            host_new_f[i * 9 + j] = new_f.at(j);
        }
        host_rho[i] = cell.getRho();
        host_ux[i] = macroU.at(0);
        host_uy[i] = macroU.at(1);
        host_boundary[i] = __calculateBoundary(boundary);
        host_obstacle[i] = obstacle;
    }

    // copy host data to device
    hipMemcpy(dev_f, host_f, cells.getTotalSize() * 9 * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_new_f, host_new_f, cells.getTotalSize() * 9 * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_rho, host_rho, cells.getTotalSize() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_ux, host_ux, cells.getTotalSize() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_uy, host_uy, cells.getTotalSize() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_boundary, host_boundary, cells.getTotalSize() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_obstacle, host_obstacle, cells.getTotalSize() * sizeof(bool), hipMemcpyHostToDevice);

    // free host memory
    hipHostFree(host_f);
    hipHostFree(host_new_f);
    hipHostFree(host_rho);
    hipHostFree(host_boundary);
    hipHostFree(host_obstacle);

    const dim3 threadsPerBlock(24, 24);
    const dim3 numBlocks(ceil(cells.getShape().at(0) / 24.0), ceil(cells.getShape().at(1) / 24.0));
    // loop
    const float temp = 2.0 * sigma * sigma;
    const float halfOmpOmmSub = 0.5 * (omP - omM);
    const float halfOmpOmmSum = 0.5 * (omP + omM);
    while (timeInstant <= maxIt)
    {
        const float uLidNow = uLid * (1.0 - std::exp(-static_cast<double>(timeInstant * timeInstant) / temp));
        GpuSimulation::step1<<<numBlocks, threadsPerBlock>>>(nx, ny, timeInstant, problemType, uLidNow, omP, halfOmpOmmSum, halfOmpOmmSub, dev_f,
                                                             dev_new_f, dev_rho, dev_ux, dev_uy, dev_boundary,
                                                             dev_obstacle);
        GpuSimulation::step2<<<numBlocks, threadsPerBlock>>>(nx, ny, dev_f, dev_new_f, dev_boundary, dev_obstacle);

        // write to file every maxIt/100 time steps
        if (timeInstant % (maxIt / 100) == 0)
        {
            // copy ux and uy to host
            hipMemcpy(host_ux, dev_ux, cells.getTotalSize() * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(host_uy, dev_uy, cells.getTotalSize() * sizeof(float), hipMemcpyDeviceToHost);
            // write to file time instant
            file << timeInstant << '\n';

            // write ux
                            for (int i = 0; i < cells.getTotalSize(); ++i)
                {
                    file << host_ux[i] << ' ';
                            }
            file << '\n';
            // write uy
                            for (int i = 0; i < cells.getTotalSize(); ++i)
                {
                    file << host_uy[i] << ' ';
                            }
            file << '\n';
            // print to console
            std::cout << "Time step: " << timeInstant << '\n';
        }

        // advance time
        timeInstant++;
    }
    hipHostFree(host_ux);
    hipHostFree(host_uy);
}

Cell &Lattice::getCellAtIndices(const std::vector<int> &indices)
{
    return cells.getElement(indices);
}

Cell &Lattice::getCellAtIndices(const int x, const int y)
{
    return cells.getElement(x, y);
}

Cell &Lattice::getCellAtIndices(const int *indices)
{
    if (structure.dimensions == 2)
    {
        return cells.getElement(indices[0], indices[1]);
    }
    else if (structure.dimensions == 3)
    {
        return cells.getElement(indices[0], indices[1], indices[2]);
    }
    else
    {
        throw std::runtime_error("Invalid number of dimensions");
    }
}

Cell &Lattice::getCellAtIndices(const int x, const int y, const int z)
{
    return cells.getElement(x, y, z);
}

const std::vector<int> Lattice::getShape() const
{
    return cells.getShape();
}

bool Lattice::isLid() const
{
    return problemType == 1;
}

const Structure &Lattice::getStructure() const
{
    return structure;
}