#include "hip/hip_runtime.h"
#include "GpuSimulation.cuh"

__device__ void zouHe(float *f, float *new_f, float *rho, float *ux, float *uy, int *boundary)
{
    return;
}

__device__ void updateMacro(float *f, float *new_f, float *rho, float *ux, float *uy, int *boundary,
                            const int *velocitiesX, const int *velocitiesY)
{
    *rho = 0;
    *ux = 0;
    *uy = 0;
    for (int i = 0; i < 9; i++)
    {
        *rho += f[i];
        *ux += f[i] * velocitiesX[i];
        *uy += f[i] * velocitiesY[i];
    }
}

__device__ void setInlets(float *ux, float *uy, const int problem_type, const float u_lid, const int row, const int col,
                          const int nx, const int ny)
{
    if (problem_type == 1)
    {
        // if i'm a any boundary set ux uy to 0
        if (row == 0 || row == ny - 1 || col == 0 || col == nx - 1)
        {
            *ux = 0;
            *uy = 0;
        }
        // if i'm on the lid set ux to u_lid
        if (row == 0)
        {
            *ux = u_lid;
        }
    }
}

__device__ void equilibriumCollision(float *f, float *new_f, float *ux, float *uy, const int *velocitiesX,
                                     const int *velocitiesY, const float *weights, const float om_p, const float om_m)
{
    return;
}

__global__ void GpuSimulation::step1(const int nx, const int ny, const int it, const int problem_type,
                                     const float u_lid, const float om_p, const float om_m, float *f, float *new_f,
                                     float *rho, float *ux, float *uy, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};
    const int index = row * nx + col;
    const int index9 = index * 9;

    // zouhe if not it 0
    if (it != 0)
    {
        zouHe(&f[index9], &new_f[index9], &rho[index], &ux[index], &uy[index], &boundary[index]);
    }

    // update macro
    updateMacro(&f[index9], &new_f[index9], &rho[index], &ux[index], &uy[index], &boundary[index], velocitiesX,
                velocitiesY);

    // set inlets
    setInlets(&ux[index], &uy[index], problem_type, u_lid, row, col, nx, ny);

    // equilibrium and collision
}

__global__ void GpuSimulation::step2(const int nx, const int ny, float *f, float *new_f, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
}
