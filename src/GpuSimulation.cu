#include "hip/hip_runtime.h"
#include "GpuSimulation.cuh"
#include "Lattice.hpp"
#include <iostream>

__device__ void step1dev(const int nx, const int ny, const int it, const int problem_type, const float u_lid,
                         const float om_p, const float halfOmpOmmSum, const float halfOmpOmmSub, const int row,
                         const int col, float *f, float *new_f, float *rho, float &ux, float &uy, const int *boundary)
{
    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};
    const int opposite[9] = {0, 3, 4, 1, 2, 7, 8, 5, 6};
    const int index = row * nx + col;
    float &rho_here = rho[index];

    // if i'm a any boundary set u to 0
    if (row == 0 || row == ny - 1 || col == 0 || col == nx - 1)
    {
        ux = 0;
        uy = 0;
    }

    // set lid inlet
    if (problem_type == 1 && row == 0)
    {
        ux = u_lid;
    }
    // set parabolic profile inlet
    else if (problem_type == 2 && col == 0)
    {
        const float halfDim = static_cast<float>(ny - 1) / 2.0;
        const float temp = static_cast<float>(row / halfDim) - 1.0;
        const float mul = 1.0 - temp * temp;
        ux = u_lid * mul;
    }

    // zouHe

    // top wall
    if (row == 0 && col != 0 && col != nx - 1)
    {
        rho_here = (f[0] + f[1] + f[3] + 2.0 * (f[2] + f[5] + f[6])) / (1.0 + uy);
        f[4] = f[2] - 2.0 / 3.0 * rho_here * uy;
        f[7] = f[5] + 0.5 * (f[1] - f[3]) - 0.5 * rho_here * ux - 1.0 / 6.0 * rho_here * uy;
        f[8] = f[6] - 0.5 * (f[1] - f[3]) + 0.5 * rho_here * ux - 1.0 / 6.0 * rho_here * uy;
    }
    // right wall
    else if (col == nx - 1 && row != 0 && row != ny - 1)
    {
        if (problem_type == 1)
        {
            rho_here = (f[0] + f[2] + f[4] + 2.0 * (f[1] + f[5] + f[8])) / (1.0 + ux);
            f[3] = f[1] - 2.0 / 3.0 * rho_here * ux;
            f[6] = f[8] - 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho_here * ux + 0.5 * rho_here * uy;
            f[7] = f[5] + 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho_here * ux - 0.5 * rho_here * uy;
        }
        else if (problem_type == 2)
        {
            rho_here = 1;
            ux = f[0] + f[2] + f[4] + 2.0 * (f[1] + f[5] + f[8]) - 1.0;
            f[3] = f[1] - 2.0 / 3.0 * ux;
            f[6] = f[8] - 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * ux;
            f[7] = f[5] + 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * ux;
        }
    }
    // bottom wall
    else if (row == ny - 1 && col != 0 && col != nx - 1)
    {
        rho_here = (f[0] + f[1] + f[3] + 2.0 * (f[4] + f[7] + f[8])) / (1.0 - uy);
        f[2] = f[4] + 2.0 / 3.0 * rho_here * uy;
        f[5] = f[7] - 0.5 * (f[1] - f[3]) + 0.5 * rho_here * ux + 1.0 / 6.0 * rho_here * uy;
        f[6] = f[8] + 0.5 * (f[1] - f[3]) - 0.5 * rho_here * ux + 1.0 / 6.0 * rho_here * uy;
    }
    // left wall
    else if (col == 0 && row != 0 && row != ny - 1)
    {
        rho_here = (f[0] + f[2] + f[4] + 2.0 * (f[3] + f[7] + f[6])) / (1.0 - ux);
        f[1] = f[3] + 2.0 / 3.0 * rho_here * ux;
        f[5] = f[7] - 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho_here * ux + 0.5 * rho_here * uy;
        f[8] = f[6] + 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho_here * ux - 0.5 * rho_here * uy;
    }
    // top right corner
    else if (row == 0 && col == nx - 1)
    {
        rho_here = rho[index - 1];
        f[3] = f[1] - 2.0 / 3.0 * rho_here * ux;
        f[4] = f[2] - 2.0 / 3.0 * rho_here * uy;
        f[7] = f[5] - 1.0 / 6.0 * rho_here * ux - 1.0 / 6.0 * rho_here * uy;
        f[8] = 0;
        f[6] = 0;
        f[0] = rho_here - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
    }
    // bottom right corner
    else if (row == ny - 1 && col == nx - 1)
    {
        rho_here = rho[index - 1];
        f[3] = f[1] - 2.0 / 3.0 * rho_here * ux;
        f[2] = f[4] + 2.0 / 3.0 * rho_here * uy;
        f[6] = f[8] + 1.0 / 6.0 * rho_here * uy - 1.0 / 6.0 * rho_here * ux;
        f[7] = 0;
        f[5] = 0;
        f[0] = rho_here - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
    }
    // bottom left corner
    else if (row == ny - 1 && col == 0)
    {
        rho_here = rho[index + 1];
        f[1] = f[3] + 2.0 / 3.0 * rho_here * ux;
        f[2] = f[4] + 2.0 / 3.0 * rho_here * uy;
        f[5] = f[7] + 1.0 / 6.0 * rho_here * ux + 1.0 / 6.0 * rho_here * uy;
        f[6] = 0;
        f[8] = 0;
        f[0] = rho_here - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
    }
    // top left corner
    else if (row == 0 && col == 0)
    {
        rho_here = rho[index + 1];
        f[1] = f[3] + 2.0 / 3.0 * rho_here * ux;
        f[4] = f[2] - 2.0 / 3.0 * rho_here * uy;
        f[8] = f[6] + 1.0 / 6.0 * rho_here * ux - 1.0 / 6.0 * rho_here * uy;
        f[7] = 0;
        f[5] = 0;
        f[0] = rho_here - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
    }

    // update macro
    rho_here = 0;
    ux = 0;
    uy = 0;
    for (int i = 0; i < 9; i++)
    {
        rho_here += f[i];
        ux += f[i] * velocitiesX[i];
        uy += f[i] * velocitiesY[i];
    }
    ux /= rho_here;
    uy /= rho_here;

    // equilibrium
    float feq[9];
    const float temp1 = 1.5 * (ux * ux + uy * uy);
    for (int i = 0; i < 9; i++)
    {
        const float temp2 = 3.0 * (velocitiesX[i] * ux + velocitiesY[i] * uy);
        feq[i] = weights[i] * rho_here * (1.0 + temp2 + 0.5 * temp2 * temp2 - temp1);
    }

    // collision for index 0
    new_f[0] = (1.0 - om_p) * f[0] + om_p * feq[0];

    // collision for other indices
    for (int i = 1; i < 9; i++)
    {
        new_f[i] = (1.0 - halfOmpOmmSum) * f[i] - halfOmpOmmSub * f[opposite[i]] + halfOmpOmmSum * feq[i] +
                   halfOmpOmmSub * feq[opposite[i]];
    }

    if (problem_type == 2)
    {
        // regular bounce back
        if (boundary[0] == 1)
        {
            f[3] = new_f[1];
        }
        else if (boundary[0] == -1)
        {
            f[1] = new_f[3];
        }
        if (boundary[1] == 1)
        {
            f[2] = new_f[4];
        }
        else if (boundary[1] == -1)
        {
            f[4] = new_f[2];
        }
        if (boundary[2] == 1)
        {
            f[6] = new_f[8];
        }
        else if (boundary[2] == -1)
        {
            f[8] = new_f[6];
        }
        if (boundary[3] == 1)
        {
            f[5] = new_f[7];
        }
        else if (boundary[3] == -1)
        {
            f[7] = new_f[5];
        }
    }
}

__global__ void step1(const int nx, const int ny, const int it, const int problem_type, const float u_lid,
                      const float om_p, const float halfOmpOmmSum, const float halfOmpOmmSub, float *f, float *new_f,
                      float *rho, float *ux, float *uy, const int *boundary, const bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int index = row * nx + col;
    const int index9 = index * 9;
    const int index4 = index * 4;

    step1dev(nx, ny, it, problem_type, u_lid, om_p, halfOmpOmmSum, halfOmpOmmSub, row, col, &f[index9], &new_f[index9],
             rho, ux[index], uy[index], &boundary[index4]);
}

__global__ void step2(const int nx, const int ny, float *f, const float *new_f, const int *boundary,
                      const bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};

    const int index = row * nx + col;
    const int index9 = index * 9;

    // stream for index 0
    f[index9] = new_f[index9];

    // stream for other indices
    for (int i = 1; i < 9; i++)
    {
        // obtain new indices
        const int new_row = row + velocitiesY[i];
        const int new_col = col + velocitiesX[i];
        const int new_index = new_row * nx + new_col;
        // stream if new index is not out of bounds or obstacle
        if (new_row >= 0 && new_row < ny && new_col >= 0 && new_col < nx && !obstacle[new_index])
        {
            const int new_index9 = new_index * 9;
            f[new_index9 + i] = new_f[index9 + i];
        }
    }
}

__global__ void calculateLiftAndDragKernel(float *lift, float *drag, const float *f, const float *new_f,
                                           const int *boundary, const bool *obstacle, const int nx, const int ny)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int index = row * nx + col;
    const int index4 = index * 4;
    const int index9 = index * 9;
    const int *boundary_here = &boundary[index4];
    const float *f_here = &f[index9];
    const float *new_f_here = &new_f[index9];

    // Calculate lift and drag for this thread's data
    const float adj = -2.0 / 0.04;
    float localLift = 0;
    float localDrag = 0;

    if (boundary_here[0] == 1)
    {
        localDrag += new_f_here[1] + f_here[3];
    }
    else if (boundary_here[0] == -1)
    {
        localDrag -= new_f_here[3] + f_here[1];
    }
    if (boundary_here[1] == 1)
    {
        localLift += new_f_here[4] + f_here[2];
    }
    else if (boundary_here[1] == -1)
    {
        localLift -= new_f_here[2] + f_here[4];
    }
    if (boundary_here[2] == 1)
    {
        localDrag += new_f_here[8] + f_here[6];
        localLift += new_f_here[8] + f_here[6];
    }
    else if (boundary_here[2] == -1)
    {
        localDrag -= new_f_here[6] + f_here[8];
        localLift -= new_f_here[6] + f_here[8];
    }
    if (boundary_here[3] == 1)
    {
        localDrag -= new_f_here[7] + f_here[5];
        localLift += new_f_here[7] + f_here[5];
    }
    else if (boundary_here[3] == -1)
    {
        localDrag += new_f_here[5] + f_here[7];
        localLift -= new_f_here[5] + f_here[7];
    }

    // Atomic add to global lift and drag
    atomicAdd(lift, localLift * adj);
    atomicAdd(drag, localDrag * adj);
}

void GpuSimulation::cudaCaller(const NDimensionalMatrix<Cell> &cells, const float sigma, const float omP,
                               const float omM, const int maxIt, const float uLid, const int problemType,
                               const int plotSteps, std::ofstream &velocity_out, std::ofstream &lift_drag_out)
{
    const int nx = cells.getShape().at(0);
    const int ny = cells.getShape().at(1);
    const int totalSize = nx * ny;
    int timeInstant = 0;

    // device allocations

    float *host_f, *host_new_f, *host_rho, *host_ux, *host_uy, *dev_f, *dev_new_f, *dev_rho, *dev_ux, *dev_uy;
    int *host_boundary, *dev_boundary;
    bool *host_obstacle, *dev_obstacle;

    // allocate memory on device
    hipMalloc((void **)&dev_f, totalSize * 9 * sizeof(float));
    hipMalloc((void **)&dev_new_f, totalSize * 9 * sizeof(float));
    hipMalloc((void **)&dev_rho, totalSize * sizeof(float));
    hipMalloc((void **)&dev_ux, totalSize * sizeof(float));
    hipMalloc((void **)&dev_uy, totalSize * sizeof(float));
    hipMalloc((void **)&dev_boundary, totalSize * 4 * sizeof(int));
    hipMalloc((void **)&dev_obstacle, totalSize * sizeof(bool));

    // allocate memory on host
    hipHostMalloc((void **)&host_f, totalSize * 9 * sizeof(float));
    hipHostMalloc((void **)&host_new_f, totalSize * 9 * sizeof(float));
    hipHostMalloc((void **)&host_rho, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_ux, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_uy, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_boundary, totalSize * 4 * sizeof(int));
    hipHostMalloc((void **)&host_obstacle, totalSize * sizeof(bool));

    // set host data
    for (int i = 0; i < totalSize; ++i)
    {
        const Cell &cell = cells.getElementAtFlatIndex(i);
        const std::vector<float> &f = cell.getF();
        const std::vector<float> &new_f = cell.getNewF();
        const std::vector<float> &macroU = cell.getMacroU();
        const std::vector<int> &boundary = cell.getBoundary();
        const bool &obstacle = cell.isObstacle();

        for (int j = 0; j < 9; ++j)
        {
            host_f[i * 9 + j] = f.at(j);
            host_new_f[i * 9 + j] = new_f.at(j);
        }
        host_rho[i] = cell.getRho();
        host_ux[i] = macroU.at(0);
        host_uy[i] = macroU.at(1);
        for (int j = 0; j < 4; ++j)
        {
            host_boundary[i * 4 + j] = boundary.at(j);
        }
        host_obstacle[i] = obstacle;
    }

    // copy host data to device
    hipMemcpy(dev_f, host_f, totalSize * 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_new_f, host_new_f, totalSize * 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_rho, host_rho, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_ux, host_ux, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_uy, host_uy, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_boundary, host_boundary, totalSize * 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_obstacle, host_obstacle, totalSize * sizeof(bool), hipMemcpyHostToDevice);

    // free host memory
    hipHostFree(host_f);
    hipHostFree(host_new_f);
    hipHostFree(host_rho);
    hipHostFree(host_boundary);
    hipHostFree(host_obstacle);

    // variables for lift and drag
    float drag, lift;
    float *dev_drag, *dev_lift;
    hipMalloc((void **)&dev_drag, sizeof(float));
    hipMalloc((void **)&dev_lift, sizeof(float));

    const dim3 threadsPerBlock(24, 24);
    const dim3 numBlocks(ceil(nx / 24.0), ceil(ny / 24.0));
    // loop
    const float temp = 2.0 * sigma * sigma;
    const float halfOmpOmmSub = 0.5 * (omP - omM);
    const float halfOmpOmmSum = 0.5 * (omP + omM);
    while (timeInstant <= maxIt)
    {
        const float uLidNow = uLid * (1.0 - std::exp(-static_cast<double>(timeInstant * timeInstant) / temp));
        step1<<<numBlocks, threadsPerBlock>>>(nx, ny, timeInstant, problemType, uLidNow, omP, halfOmpOmmSum,
                                              halfOmpOmmSub, dev_f, dev_new_f, dev_rho, dev_ux, dev_uy, dev_boundary,
                                              dev_obstacle);
        step2<<<numBlocks, threadsPerBlock>>>(nx, ny, dev_f, dev_new_f, dev_boundary, dev_obstacle);

        if (problemType == 2 && timeInstant % (maxIt / plotSteps) == 0)
        {
            // clear lift and drag
            hipMemset(dev_lift, 0, sizeof(float));
            hipMemset(dev_drag, 0, sizeof(float));
            // Launch kernel
            calculateLiftAndDragKernel<<<numBlocks, threadsPerBlock>>>(dev_lift, dev_drag, dev_f, dev_new_f,
                                                                       dev_boundary, dev_obstacle, nx, ny);

            // Copy results back to host
            hipMemcpy(&lift, dev_lift, sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&drag, dev_drag, sizeof(float), hipMemcpyDeviceToHost);
        }

        // write to file every maxIt/plotSteps time steps
        if (timeInstant % (maxIt / plotSteps) == 0)
        {
            // copy ux and uy to host
            hipMemcpy(host_ux, dev_ux, totalSize * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(host_uy, dev_uy, totalSize * sizeof(float), hipMemcpyDeviceToHost);

            // write to files time instant
            velocity_out << timeInstant << '\n';
            lift_drag_out << timeInstant << '\n';

            // write ux
            for (int i = 0; i < totalSize; ++i)
            {
                velocity_out << host_ux[i] << ' ';
            }
            velocity_out << '\n';
            // write uy
            for (int i = 0; i < totalSize; ++i)
            {
                velocity_out << host_uy[i] << ' ';
            }
            velocity_out << '\n';

            // lift and drag
            if (problemType == 2)
            {
                lift_drag_out << drag << ' ' << lift << '\n';
            }

            // print to console
            std::cout << "Time step: " << timeInstant << '\n';
        }

        // advance time
        timeInstant++;
    }
    hipHostFree(host_ux);
    hipHostFree(host_uy);
}
