#include "hip/hip_runtime.h"
#include "GpuSimulation.cuh"

__device__ void step1dev(const int nx, const int ny, const int it, const int problem_type, const float u_lid,
                         const float om_p, const float halfOmpOmmSum, const float halfOmpOmmSub, const int row,
                         const int col, float *f, float *new_f, float &rho, float &ux, float &uy, int &boundary)
{
    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};
    const int opposite[9] = {0, 3, 4, 1, 2, 7, 8, 5, 6};

    // zouHe if it != 0
    if (it != 0)
    {
        // top wall
        if (boundary == 2)
        {
            rho = (f[0] + f[1] + f[3] + 2.0 * (f[2] + f[5] + f[6])) / (1.0 + uy);
            f[4] = f[2] - 2.0 / 3.0 * rho * uy;
            f[7] = f[5] + 0.5 * (f[1] - f[3]) - 0.5 * rho * ux - 1.0 / 6.0 * rho * uy;
            f[8] = f[6] - 0.5 * (f[1] - f[3]) + 0.5 * rho * ux - 1.0 / 6.0 * rho * uy;
        }
        // right wall
        else if (boundary == 1)
        {
            rho = (f[0] + f[2] + f[4] + 2.0 * (f[1] + f[5] + f[8])) / (1.0 + ux);
            f[3] = f[1] - 2.0 / 3.0 * rho * ux;
            f[6] = f[8] - 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho * ux + 0.5 * rho * uy;
            f[7] = f[5] + 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho * ux - 0.5 * rho * uy;
        }
        // bottom wall
        else if (boundary == 4)
        {
            rho = (f[0] + f[1] + f[3] + 2.0 * (f[4] + f[7] + f[8])) / (1.0 - uy);
            f[2] = f[4] + 2.0 / 3.0 * rho * uy;
            f[5] = f[7] - 0.5 * (f[1] - f[3]) + 0.5 * rho * ux + 1.0 / 6.0 * rho * uy;
            f[6] = f[8] + 0.5 * (f[1] - f[3]) - 0.5 * rho * ux + 1.0 / 6.0 * rho * uy;
        }
        // left wall
        else if (boundary == 3)
        {
            rho = (f[0] + f[2] + f[4] + 2.0 * (f[3] + f[6] + f[7])) / (1.0 - ux);
            f[1] = f[3] - 2.0 / 3.0 * rho * ux;
            f[5] = f[7] - 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho * ux + 0.5 * rho * uy;
            f[8] = f[6] + 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho * ux - 0.5 * rho * uy;
        }
        // top right corner
        else if (boundary == 5)
        {
            f[3] = f[1] - 2.0 / 3.0 * rho * ux;
            f[4] = f[2] - 2.0 / 3.0 * rho * uy;
            f[7] = f[5] - 1.0 / 6.0 * rho * ux - 1.0 / 6.0 * rho * uy;
            f[8] = 0;
            f[6] = 0;
            f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
        }
        // bottom right corner
        else if (boundary == 8)
        {
            f[3] = f[1] - 2.0 / 3.0 * rho * ux;
            f[2] = f[4] + 2.0 / 3.0 * rho * uy;
            f[6] = f[8] + 1.0 / 6.0 * rho * uy - 1.0 / 6.0 * rho * ux;
            f[7] = 0;
            f[5] = 0;
            f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
        }
        // bottom left corner
        else if (boundary == 7)
        {
            f[1] = f[3] + 2.0 / 3.0 * rho * ux;
            f[2] = f[4] + 2.0 / 3.0 * rho * uy;
            f[5] = f[7] + 1.0 / 6.0 * rho * ux + 1.0 / 6.0 * rho * uy;
            f[6] = 0;
            f[8] = 0;
            f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
        }
        // top left corner
        else if (boundary == 6)
        {
            f[1] = f[3] + 2.0 / 3.0 * rho * ux;
            f[4] = f[2] - 2.0 / 3.0 * rho * uy;
            f[8] = f[6] - 1.0 / 6.0 * rho * ux + 1.0 / 6.0 * rho * uy;
            f[7] = 0;
            f[5] = 0;
            f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
        }
    }

    // update macro
    rho = 0;
    ux = 0;
    uy = 0;
    for (int i = 0; i < 9; i++)
    {
        rho += f[i];
        ux += f[i] * velocitiesX[i];
        uy += f[i] * velocitiesY[i];
    }
    ux /= rho;
    uy /= rho;

    // set inlets
    if (problem_type == 1)
    {
        // if i'm a any boundary set ux uy to 0
        if (row == 0 || row == ny - 1 || col == 0 || col == nx - 1)
        {
            ux = 0;
            uy = 0;
        }
        // if i'm on the lid set ux to u_lid
        if (row == 0)
        {
            ux = u_lid;
        }
    }

    // equilibrium
    float feq[9];
    const float temp1 = 1.5 * (ux * ux + uy * uy);
    for (int i = 0; i < 9; i++)
    {
        const float temp2 = 3.0 * (velocitiesX[i] * ux + velocitiesY[i] * uy);
        feq[i] = weights[i] * rho * (1.0 + temp2 + 0.5 * temp2 * temp2 - temp1);
    }

    // collision for index 0
    new_f[0] = (1.0 - om_p) * f[0] + om_p * feq[0];

    // collision for other indices
    for (int i = 1; i < 9; i++)
    {
        new_f[i] = (1.0 - halfOmpOmmSum) * f[i] - halfOmpOmmSub * f[opposite[i]] + halfOmpOmmSum * feq[i] +
                   halfOmpOmmSub * feq[opposite[i]];
    }
}

__global__ void GpuSimulation::step1(const int nx, const int ny, const int it, const int problem_type,
                                     const float u_lid, const float om_p, const float halfOmpOmmSum,
                                     const float halfOmpOmmSub, float *f, float *new_f, float *rho, float *ux,
                                     float *uy, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int index = row * nx + col;
    const int index9 = index * 9;

    step1dev(nx, ny, it, problem_type, u_lid, om_p, halfOmpOmmSum, halfOmpOmmSub, row, col, &f[index9], &new_f[index9],
             rho[index], ux[index], uy[index], boundary[index]);
}

__global__ void GpuSimulation::step2(const int nx, const int ny, float *f, float *new_f, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};

    const int index = row * nx + col;
    const int index9 = index * 9;

    // stream for index 0
    f[index9] = new_f[index9];

    const int boundary_here = boundary[index];
    // stream for other indices
    for (int i = 1; i < 9; i++)
    {
        // check if there's a boundary in the way
        if ((velocitiesX[boundary_here] != velocitiesX[i] || velocitiesX[boundary_here] == 0) &&
            (velocitiesY[boundary_here] != velocitiesY[i] || velocitiesY[boundary_here] == 0))
        {
            // obtain new cell coordinates
            const int new_index9 = (row + velocitiesY[i]) * nx * 9 + (col + velocitiesX[i]) * 9;
            // stream
            f[new_index9 + i] = new_f[index9 + i];
        }
    }
}
