#include "hip/hip_runtime.h"
#include "GpuSimulation.cuh"
#include "Lattice.hpp"
#include <iostream>

__device__ void step1dev(const int nx, const int ny, const int it, const int problem_type, const float u_lid,
                         const float om_p, const float halfOmpOmmSum, const float halfOmpOmmSub, const int row,
                         const int col, float *f, float *new_f, float &rho, float &ux, float &uy, int *boundary)
{
    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};
    const float weights[9] = {4.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0,  1.0 / 9.0, 1.0 / 9.0,
                              1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};
    const int opposite[9] = {0, 3, 4, 1, 2, 7, 8, 5, 6};

    // if i'm a any boundary set u to 0
    if (row == 0 || row == ny - 1 || col == 0 || col == nx - 1)
    {
        ux = 0;
        uy = 0;
    }

    // set inlets
    if (problem_type == 1)
    {
        // if i'm on the lid set ux to u_lid
        if (row == 0)
        {
            ux = u_lid;
        }
    }
    else if (problem_type == 2)
    {
        // parabolic profile
        const float halfDim = static_cast<float>(ny - 1) / 2.0;
        const float temp = static_cast<float>(row / halfDim) - 1.0;
        const float mul = 1.0 - temp * temp;
        ux = u_lid * mul;
    }

    // zouHe

    // top wall
    if (row == 0 && col != 0 && col != nx - 1)
    {
        rho = (f[0] + f[1] + f[3] + 2.0 * (f[2] + f[5] + f[6])) / (1.0 + uy);
        f[4] = f[2] - 2.0 / 3.0 * rho * uy;
        f[7] = f[5] + 0.5 * (f[1] - f[3]) - 0.5 * rho * ux - 1.0 / 6.0 * rho * uy;
        f[8] = f[6] - 0.5 * (f[1] - f[3]) + 0.5 * rho * ux - 1.0 / 6.0 * rho * uy;
    }
    // right wall
    else if (col == nx - 1 && row != 0 && row != ny - 1)
    {
        if (problem_type == 1)
        {
            rho = (f[0] + f[2] + f[4] + 2.0 * (f[1] + f[5] + f[8])) / (1.0 + ux);
            f[3] = f[1] - 2.0 / 3.0 * rho * ux;
            f[6] = f[8] - 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho * ux + 0.5 * rho * uy;
            f[7] = f[5] + 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * rho * ux - 0.5 * rho * uy;
        }
        else if (problem_type == 2)
        {
            rho = 1;
            ux = f[0] + f[2] + f[4] + 2.0 * (f[1] + f[5] + f[8]) - 1.0;
            f[3] = f[1] - 2.0 / 3.0 * ux;
            f[6] = f[8] - 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * ux;
            f[7] = f[5] + 0.5 * (f[2] - f[4]) - 1.0 / 6.0 * ux;
        }
    }
    // bottom wall
    else if (row == ny - 1 && col != 0 && col != nx - 1)
    {
        rho = (f[0] + f[1] + f[3] + 2.0 * (f[4] + f[7] + f[8])) / (1.0 - uy);
        f[2] = f[4] + 2.0 / 3.0 * rho * uy;
        f[5] = f[7] - 0.5 * (f[1] - f[3]) + 0.5 * rho * ux + 1.0 / 6.0 * rho * uy;
        f[6] = f[8] + 0.5 * (f[1] - f[3]) - 0.5 * rho * ux + 1.0 / 6.0 * rho * uy;
    }
    // left wall
    else if (col == 0 && row != 0 && row != ny - 1)
    {
        rho = (f[0] + f[2] + f[4] + 2.0 * (f[3] + f[6] + f[7])) / (1.0 - ux);
        f[1] = f[3] - 2.0 / 3.0 * rho * ux;
        f[5] = f[7] - 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho * ux + 0.5 * rho * uy;
        f[8] = f[6] + 0.5 * (f[2] - f[4]) + 1.0 / 6.0 * rho * ux - 0.5 * rho * uy;
    }
    // top right corner
    else if (row == 0 && col == nx - 1)
    {
        f[3] = f[1] - 2.0 / 3.0 * rho * ux;
        f[4] = f[2] - 2.0 / 3.0 * rho * uy;
        f[7] = f[5] - 1.0 / 6.0 * rho * ux - 1.0 / 6.0 * rho * uy;
        f[8] = 0;
        f[6] = 0;
        f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
    }
    // bottom right corner
    else if (row == ny - 1 && col == nx - 1)
    {
        f[3] = f[1] - 2.0 / 3.0 * rho * ux;
        f[2] = f[4] + 2.0 / 3.0 * rho * uy;
        f[6] = f[8] + 1.0 / 6.0 * rho * uy - 1.0 / 6.0 * rho * ux;
        f[7] = 0;
        f[5] = 0;
        f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
    }
    // bottom left corner
    else if (row == ny - 1 && col == 0)
    {
        f[1] = f[3] + 2.0 / 3.0 * rho * ux;
        f[2] = f[4] + 2.0 / 3.0 * rho * uy;
        f[5] = f[7] + 1.0 / 6.0 * rho * ux + 1.0 / 6.0 * rho * uy;
        f[6] = 0;
        f[8] = 0;
        f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[5] - f[7];
    }
    // top left corner
    else if (row == 0 && col == 0)
    {
        f[1] = f[3] + 2.0 / 3.0 * rho * ux;
        f[4] = f[2] - 2.0 / 3.0 * rho * uy;
        f[8] = f[6] - 1.0 / 6.0 * rho * ux + 1.0 / 6.0 * rho * uy;
        f[7] = 0;
        f[5] = 0;
        f[0] = rho - f[1] - f[2] - f[3] - f[4] - f[6] - f[8];
    }

    // update macro
    rho = 0;
    ux = 0;
    uy = 0;
    for (int i = 0; i < 9; i++)
    {
        rho += f[i];
        ux += f[i] * velocitiesX[i];
        uy += f[i] * velocitiesY[i];
    }
    ux /= rho;
    uy /= rho;

    // equilibrium
    float feq[9];
    const float temp1 = 1.5 * (ux * ux + uy * uy);
    for (int i = 0; i < 9; i++)
    {
        const float temp2 = 3.0 * (velocitiesX[i] * ux + velocitiesY[i] * uy);
        feq[i] = weights[i] * rho * (1.0 + temp2 + 0.5 * temp2 * temp2 - temp1);
    }

    // collision for index 0
    new_f[0] = (1.0 - om_p) * f[0] + om_p * feq[0];

    // collision for other indices
    for (int i = 1; i < 9; i++)
    {
        new_f[i] = (1.0 - halfOmpOmmSum) * f[i] - halfOmpOmmSub * f[opposite[i]] + halfOmpOmmSum * feq[i] +
                   halfOmpOmmSub * feq[opposite[i]];
    }

    if (problem_type == 2)
    {
        // regular bounce back
        if (boundary[0] == 1)
        {
            f[3] = new_f[1];
        }
        else if (boundary[0] == -1)
        {
            f[1] = new_f[3];
        }
        if (boundary[1] == 1)
        {
            f[2] = new_f[4];
        }
        else if (boundary[1] == -1)
        {
            f[4] = new_f[2];
        }
        if (boundary[2] == 1)
        {
            f[6] = new_f[8];
        }
        else if (boundary[2] == -1)
        {
            f[8] = new_f[6];
        }
        if (boundary[3] == 1)
        {
            f[5] = new_f[7];
        }
        else if (boundary[3] == -1)
        {
            f[7] = new_f[5];
        }
    }
}

__global__ void step1(const int nx, const int ny, const int it, const int problem_type, const float u_lid,
                      const float om_p, const float halfOmpOmmSum, const float halfOmpOmmSub, float *f, float *new_f,
                      float *rho, float *ux, float *uy, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int index = row * nx + col;
    const int index9 = index * 9;
    const int index4 = index * 4;

    step1dev(nx, ny, it, problem_type, u_lid, om_p, halfOmpOmmSum, halfOmpOmmSub, row, col, &f[index9], &new_f[index9],
             rho[index], ux[index], uy[index], &boundary[index4]);
}

__global__ void step2(const int nx, const int ny, float *f, float *new_f, int *boundary, bool *obstacle)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    // return if out of bounds or obstacle
    if (row >= ny || col >= nx || obstacle[row * nx + col])
        return;

    const int velocitiesX[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
    const int velocitiesY[9] = {0, 0, -1, 0, 1, -1, -1, 1, 1};

    const int index = row * nx + col;
    const int index9 = index * 9;

    // stream for index 0
    f[index9] = new_f[index9];

    // stream for other indices
    for (int i = 1; i < 9; i++)
    {
        // obtain new indices
        const int new_row = row + velocitiesY[i];
        const int new_col = col + velocitiesX[i];
        const int new_index = new_row * nx + new_col;
        // stream if new index is not out of bounds or obstacle
        if (new_row >= 0 && new_row < ny && new_col >= 0 && new_col < nx && !obstacle[new_index])
        {
            const int new_index9 = new_index * 9;
            f[new_index9 + i] = new_f[index9 + i];
        }
    }
}

void GpuSimulation::cudaCaller(const NDimensionalMatrix<Cell> &cells, const float sigma, const float omP,
                               const float omM, const int maxIt, const float uLid, const int problemType,
                               const Structure &structure, std::ofstream &file, const int plotSteps)
{
    const int nx = cells.getShape().at(0);
    const int ny = cells.getShape().at(1);
    const int totalSize = nx * ny;
    int timeInstant = 0;

    // device allocations

    float *host_f, *host_new_f, *host_rho, *host_ux, *host_uy, *dev_f, *dev_new_f, *dev_rho, *dev_ux, *dev_uy;
    int *host_boundary, *dev_boundary;
    bool *host_obstacle, *dev_obstacle;

    // allocate memory on device
    hipMalloc((void **)&dev_f, totalSize * 9 * sizeof(float));
    hipMalloc((void **)&dev_new_f, totalSize * 9 * sizeof(float));
    hipMalloc((void **)&dev_rho, totalSize * sizeof(float));
    hipMalloc((void **)&dev_ux, totalSize * sizeof(float));
    hipMalloc((void **)&dev_uy, totalSize * sizeof(float));
    hipMalloc((void **)&dev_boundary, totalSize * 4 * sizeof(int));
    hipMalloc((void **)&dev_obstacle, totalSize * sizeof(bool));

    // allocate memory on host
    hipHostMalloc((void **)&host_f, totalSize * 9 * sizeof(float));
    hipHostMalloc((void **)&host_new_f, totalSize * 9 * sizeof(float));
    hipHostMalloc((void **)&host_rho, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_ux, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_uy, totalSize * sizeof(float));
    hipHostMalloc((void **)&host_boundary, totalSize * 4 * sizeof(int));
    hipHostMalloc((void **)&host_obstacle, totalSize * sizeof(bool));

    // set host data
    for (int i = 0; i < totalSize; ++i)
    {
        const Cell &cell = cells.getElementAtFlatIndex(i);
        const std::vector<float> &f = cell.getF();
        const std::vector<float> &new_f = cell.getNewF();
        const std::vector<float> &macroU = cell.getMacroU();
        const std::vector<int> &boundary = cell.getBoundary();
        const bool &obstacle = cell.isObstacle();

        for (int j = 0; j < 9; ++j)
        {
            host_f[i * 9 + j] = f.at(j);
            host_new_f[i * 9 + j] = new_f.at(j);
        }
        host_rho[i] = cell.getRho();
        host_ux[i] = macroU.at(0);
        host_uy[i] = macroU.at(1);
        for (int j = 0; j < 4; ++j)
        {
            host_boundary[i * 4 + j] = boundary.at(j);
        }
        host_obstacle[i] = obstacle;
    }

    // copy host data to device
    hipMemcpy(dev_f, host_f, totalSize * 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_new_f, host_new_f, totalSize * 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_rho, host_rho, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_ux, host_ux, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_uy, host_uy, totalSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_boundary, host_boundary, totalSize * 4 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_obstacle, host_obstacle, totalSize * sizeof(bool), hipMemcpyHostToDevice);

    // free host memory
    hipHostFree(host_f);
    hipHostFree(host_new_f);
    hipHostFree(host_rho);
    hipHostFree(host_boundary);
    hipHostFree(host_obstacle);

    const dim3 threadsPerBlock(24, 24);
    const dim3 numBlocks(ceil(nx / 24.0), ceil(ny / 24.0));
    // loop
    const float temp = 2.0 * sigma * sigma;
    const float halfOmpOmmSub = 0.5 * (omP - omM);
    const float halfOmpOmmSum = 0.5 * (omP + omM);
    while (timeInstant <= maxIt)
    {
        const float uLidNow = uLid * (1.0 - std::exp(-static_cast<double>(timeInstant * timeInstant) / temp));
        step1<<<numBlocks, threadsPerBlock>>>(nx, ny, timeInstant, problemType, uLidNow, omP, halfOmpOmmSum,
                                              halfOmpOmmSub, dev_f, dev_new_f, dev_rho, dev_ux, dev_uy, dev_boundary,
                                              dev_obstacle);
        step2<<<numBlocks, threadsPerBlock>>>(nx, ny, dev_f, dev_new_f, dev_boundary, dev_obstacle);

        // write to file every maxIt/plotSteps time steps
        if (timeInstant % (maxIt / plotSteps) == 0)
        {
            // copy ux and uy to host
            hipMemcpy(host_ux, dev_ux, totalSize * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(host_uy, dev_uy, totalSize * sizeof(float), hipMemcpyDeviceToHost);
            // write to file time instant
            file << timeInstant << '\n';

            // write ux
            for (int i = 0; i < totalSize; ++i)
            {
                file << host_ux[i] << ' ';
            }
            file << '\n';
            // write uy
            for (int i = 0; i < totalSize; ++i)
            {
                file << host_uy[i] << ' ';
            }
            file << '\n';
            // print to console
            std::cout << "Time step: " << timeInstant << '\n';
        }

        // advance time
        timeInstant++;
    }
    hipHostFree(host_ux);
    hipHostFree(host_uy);
}
